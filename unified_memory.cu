#include "hip/hip_runtime.h"

#include <time.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>

using namespace std;

struct Node{
	int key;
	int nextIdx;
	int nextLevel;
};

__global__ void assign(Node *sl, Node *data)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int k = data[i].key;
	sl[k - 1].key = k;
	sl[k - 1].nextIdx = k;
}

__global__ void connect(Node *sl, int N)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int level = 1;
	for (int i = 2; i < N+1; i = i * 2)
	{
		if (id%i == 0)
		{
			int newid = id + level*N;
			sl[newid].key = sl[id].key;
			sl[newid].nextIdx = newid + i;
			sl[newid].nextLevel = newid - N;
		}
		level++;
	}

}


struct timespec diff(timespec start, timespec end)
{
	struct timespec temp;
	if ((end.tv_nsec - start.tv_nsec) < 0) {
		temp.tv_sec = end.tv_sec - start.tv_sec - 1;
		temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
	}
	else {
		temp.tv_sec = end.tv_sec - start.tv_sec;
		temp.tv_nsec = end.tv_nsec - start.tv_nsec;
	}
	return temp;
}


void print(Node *sl, int N, int MAX_LEVEL)
{
	cout << "Index:" << endl;
	for (int i = 0; i<N*MAX_LEVEL; i++)
	{
		cout << setw(4) << sl[i].key;
		if ((i + 1) % N == 0)
			cout << endl;
	}
	printf("NextIndex:\n");
	for (int i = 0; i<N*MAX_LEVEL; i++)
	{
		cout << setw(4) << sl[i].nextIdx;
		if ((i + 1) % N == 0)
			cout << endl;
	}
	printf("NextLevel:\n");
	for (int i = 0; i<N*MAX_LEVEL; i++)
	{
	    cout << setw(4) << sl[i].nextLevel;
		if ((i + 1) % N == 0)
			cout << endl;
	}
}

int main()
{
	int N=1024*1024, MAX_LEVEL=21;
	Node *sl, *data;
	double time_used, sum=0;
	hipError_t err = hipSuccess;
    struct timespec start, end, temp;	
	int loop;
	for(MAX_LEVEL=21;MAX_LEVEL<30;MAX_LEVEL++)
	{
 	for(loop=1;loop<11;loop++)
	{
		clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
		
		err = hipMallocManaged(&sl, N * sizeof(Node) * MAX_LEVEL);
		if(err != hipSuccess)
		{
			fprintf(stderr, "Failed to malloc sl in loop %d : %s\n", loop, hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		err = hipMallocManaged(&data, N * sizeof(Node));
		if(err != hipSuccess)
		{
			fprintf(stderr, "Failed to malloc data in loop %d : %s\n", loop, hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		for (int i = 0; i < N; i++)
		{
			data[i].key =i+1;
		}
		int random = 1000;
		while (random--) {
			int i = rand() % N;
			int j = rand() % N;
			int tmp = data[i].key;
			data[i].key = data[j].key;
			data[j].key = tmp;
		}

		for (int i = 0; i < MAX_LEVEL * N; i++) {
			sl[i].key = -1;
			sl[i].nextLevel = -1;
			sl[i].nextIdx = -1;
		}
	
		int block = N/1024;
		assign <<< block, 1024 >>> (sl,data);
		err = hipGetLastError();
		if (err != hipSuccess) 
		{
            fprintf(stderr, "Failed to assign in loop %d : %s\n", loop, hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

		connect <<< block,1024 >>> (sl, N);
		err = hipGetLastError();
		
		if (err != hipSuccess) 
		{
    	    fprintf(stderr, "Failed to connect in loop %d : %s\n", loop, hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        
		hipFree(sl);
		hipFree(data);
    	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &end);
		temp = diff(start, end);
		time_used = 1000 * (temp.tv_sec + (double)temp.tv_nsec / 1000000000.0);
		sum += time_used;

		cout << "loop: " << loop << " time: " << time_used << endl;
//		cout << sum << endl;
	}
	cout << "Data:" << N << endl << "Maxlevel: " << MAX_LEVEL << endl << sum/10 << endl << endl;
	N = N *2;
	sum=0;
	}
    return 0;
}
